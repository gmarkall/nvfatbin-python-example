
#include <hip/hip_runtime.h>
// Copyright (c) 2023-2024, NVIDIA CORPORATION. All rights reserved.

extern "C" __device__ float add(float x, float y);

__global__ void add_kernel(float* xs, float *ys)
{
  add(xs[0], ys[0]);
}
